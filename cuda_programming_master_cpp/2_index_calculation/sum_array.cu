
#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void sum_array_gpu(int *a, int *b, int *c, int size)
{
       int gid = blockIdx.x * blockDim.x + threadIdx.x;

       if (gid < size)
       {
              c[gid] = a[gid] + b[gid];
       }
}

void sum_array_cpu(int *a, int *b, int *c, int size)
{
       for (int i = 0; i < 1000; i++)
       {
              c[i] = a[i] + b[i];
       }
}

bool compareArrays(const int *arr1, const int *arr2, size_t size)
{
       for (size_t i = 0; i < size; i++)
       {
              if (arr1[i] != arr2[i])
              {
                     return false;
              }
       }
       return true;
}

int main()
{
       int size = 1000;
       int block_size = 128;

       int NO_BYTES = size * sizeof(int);

       int *h_a, *h_b, *gpu_result, *h_c;

       h_a = (int *)malloc(NO_BYTES);
       h_b = (int *)malloc(NO_BYTES);
       h_c = (int *)malloc(NO_BYTES);

       gpu_result = (int *)malloc(NO_BYTES);

       for (int i = 0; i < size; i++)
       {
              h_a[i] = i;
              h_b[i] = 2 * i;
       }

       sum_array_cpu(h_a, h_b, h_c, size);

       int *d_a, *d_b, *d_c;
       hipMalloc((int **)&d_a, NO_BYTES);
       hipMalloc((int **)&d_b, NO_BYTES);
       hipMalloc((int **)&d_c, NO_BYTES);

       hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
       hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

       dim3 block(block_size);
       dim3 grid((size / block.x) + 1);

       sum_array_gpu<<<grid, block>>>(d_a, d_b, d_c, size);
       hipDeviceSynchronize(); // block the host execution until kernel function finish

       hipMemcpy(gpu_result, d_c, NO_BYTES, hipMemcpyDeviceToHost);

       // compare arrays
       if (compareArrays(h_c, gpu_result, size)){
              printf("Same result!\n");
       } else {
              printf("Different result\n");
       }

       hipFree(d_c);
       hipFree(d_b);
       hipFree(d_a);

       free(gpu_result);
       free(h_a);
       free(h_b);
       return 0;
}

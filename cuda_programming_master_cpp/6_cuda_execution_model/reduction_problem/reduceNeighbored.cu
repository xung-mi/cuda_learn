#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <stdio.h>

#define NX (1 << 16) // Giảm kích thước để tránh lỗi bộ nhớ
#define NY (1 << 16)

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
  // set thread ID
  unsigned int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n)
    return;

  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    if ((tid % (2 * stride)) == 0) {
      idata[tid] += idata[tid + stride];
    }

    __syncthreads();
  }

  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata,
                                     unsigned int n) {
  // set thread ID
  unsigned int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n)
    return;

  for (int stride = 1; stride < blockDim.x; stride *= 2) {
    int index = 2 * stride * tid;
    if (index < blockDim.x) {
      idata[index] += idata[index + stride];
    }

    __syncthreads();
  }

  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n) {
  // set thread ID
  unsigned int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n)
    return;

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      idata[tid] += idata[tid + stride];
    }

    __syncthreads();
  }

  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n) {
  // set thread ID
  unsigned int tid = threadIdx.x;
  unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x * 2;
  // unrolling 2 data blocks
  if (idx + blockDim.x < n)
    g_idata[idx] += g_idata[idx + blockDim.x];
  __syncthreads();
  // in-place reduction in global memory
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      idata[tid] += idata[tid + stride];
    }
    // synchronize within threadblock
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n) {
  // set thread ID
  unsigned int tid = threadIdx.x;
  unsigned int idx =  blockIdx.x * blockDim.x * 4 + threadIdx.x;
  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x * 4;
  // unrolling4 data blocks
  if (idx + 3* blockDim.x < n)
    g_idata[idx] += g_idata[idx + blockDim.x] + g_idata[idx + 2*blockDim.x] + g_idata[idx+3*blockDim.x];
  __syncthreads();
  // in-place reduction in global memory
  for (int stride = blockDim.x /2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      idata[tid] += idata[tid + stride];
    }
    // synchronize within threadblock
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

int recursiveReduce(int *data, int const size) {
  if (size == 1)
    return data[0];
  int const stride = size / 2;

  for (int i = 0; i < stride; i++) {
    data[i] += data[i + stride];
  }

  return recursiveReduce(data, stride);
}

int main(int argc, char **argv) {
  int size = 1 << 24;
  printf("With array size %d ", size);

  int blocksize = 512;
  if (argc > 1) {
    blocksize = atoi(argv[1]);
  }

  dim3 block(blocksize, 1);
  dim3 grid((size + block.x - 1) / block.x, 1);
  printf("grid %d block %d\n", grid.x, block.x);

  // allocate host memory
  size_t bytes = size * sizeof(int);
  int *h_idata = (int *)malloc(bytes);
  int *h_odata = (int *)malloc(grid.x * sizeof(int));
  int *tmp = (int *)malloc(bytes);

  // initialize the array
  for (int i = 0; i < size; i++) {
    // mask off high 2 bytes to force max number to 255
    h_idata[i] = (int)(rand() & 0xFF);
  }
  memcpy(tmp, h_idata, bytes);

  clock_t iStart, iElaps;
  int gpu_sum = 0;

  // allocate device memory
  int *d_idata = NULL;
  int *d_odata = NULL;
  hipMalloc((void **)&d_idata, bytes);
  hipMalloc((void **)&d_odata, grid.x * sizeof(int));

  // cpu reduction
  iStart = clock();
  int cpu_sum = recursiveReduce(tmp, size);
  iElaps = (long int)(clock() - iStart);
  printf("cpu reduce elapsed %ld ms cpu_sum: %d\n", iElaps, cpu_sum);

  // kernel 1: reduceNeighbored
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu Neighbored elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x, block.x);

  // kernel 2: reduceNeighboredLess
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceNeighboredLess<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu NeighboredL elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x, block.x);

  // kernel 3: reduceInterLeaved
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu Interleaved elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x, block.x);

  // kernel 4: reduceUnrolling2
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceUnrolling2<<<grid.x/2, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x/2 * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x/2; i++)
    gpu_sum += h_odata[i];
  printf("gpu unroll2 elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x/2, block.x);

  // kernel 5: reduceUnrolling4
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceUnrolling4<<<grid.x/4, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x/4 * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x/4; i++)
    gpu_sum += h_odata[i];
  printf("gpu unroll2 elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x/4, block.x);

  free(h_idata);
  free(h_odata);
  hipFree(d_idata);
  hipFree(d_odata);
  hipDeviceReset();

  bool bResult = (gpu_sum == cpu_sum);
  if (!bResult)
    printf("Test failed!\n");
  else {
    printf("cpu_sum = gpu_sum\n");
  }
  return EXIT_SUCCESS;
}

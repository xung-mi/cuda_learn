#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <stdio.h>

#define NX (1 << 16) // Giảm kích thước để tránh lỗi bộ nhớ
#define NY (1 << 16)

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
  // set thread ID
  unsigned int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n)
    return;

  for (int stride = 1; stride < blockDim.x; stride *= 2)
  {
    if ((tid % (2 * stride)) == 0)
    {
      idata[tid] += idata[tid + stride];
    }

    __syncthreads();
  }

  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata,
                                     unsigned int n)
{
  // set thread ID
  unsigned int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n)
    return;

  for (int stride = 1; stride < blockDim.x; stride *= 2)
  {
    int index = 2 * stride * tid;
    if (index < blockDim.x)
    {
      idata[index] += idata[index + stride];
    }

    __syncthreads();
  }

  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
  // set thread ID
  unsigned int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int *idata = g_idata + blockIdx.x * blockDim.x;
  if (idx >= n)
    return;

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
  {
    if (tid < stride)
    {
      idata[tid] += idata[tid + stride];
    }

    __syncthreads();
  }

  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
  // set thread ID
  unsigned int tid = threadIdx.x;
  unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x * 2;
  // unrolling 2 data blocks
  if (idx + blockDim.x < n)
    g_idata[idx] += g_idata[idx + blockDim.x];
  __syncthreads();
  // in-place reduction in global memory
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
  {
    if (tid < stride)
    {
      idata[tid] += idata[tid + stride];
    }
    // synchronize within threadblock
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n)
{
  // set thread ID
  unsigned int tid = threadIdx.x;
  unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x * 4;
  // unrolling4 data blocks
  if (idx + 3 * blockDim.x < n)
    g_idata[idx] += g_idata[idx + blockDim.x] + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
  __syncthreads();
  // in-place reduction in global memory
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
  {
    if (tid < stride)
    {
      idata[tid] += idata[tid + stride];
    }
    // synchronize within threadblock
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarp8(int *g_idata, int *g_odata, int n)
{
  int tid = threadIdx.x;
  // global index of data element in data block
  int idx = blockIdx.x * blockDim.x * 8 + tid;

  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x * 8;

  // unrolling 8
  if (idx + 7 * blockDim.x < n)
  {
    int a1 = g_idata[idx];
    int a2 = g_idata[idx + blockDim.x];
    int a3 = g_idata[idx + 2 * blockDim.x];
    int a4 = g_idata[idx + 3 * blockDim.x];
    int b1 = g_idata[idx + 4 * blockDim.x];
    int b2 = g_idata[idx + 5 * blockDim.x];
    int b3 = g_idata[idx + 6 * blockDim.x];
    int b4 = g_idata[idx + 7 * blockDim.x];
    g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
  }
  __syncthreads();

  // in-place reduction and complete unroll
  if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
  __syncthreads();
  if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
  __syncthreads();
  if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
  __syncthreads();
  if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
  __syncthreads();

  // unrolling warp
  if (tid < 32)
  {
    volatile int *vsmem = idata;
    vsmem[tid] += vsmem[tid + 32];
    vsmem[tid] += vsmem[tid + 16];
    vsmem[tid] += vsmem[tid + 8];
    vsmem[tid] += vsmem[tid + 4];
    vsmem[tid] += vsmem[tid + 2];
    vsmem[tid] += vsmem[tid + 1];
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

template <unsigned int iBlockSize>
__global__ void reduceCompleteUnrollTemplate(int *g_idata, int *g_odata, int n)
{
  int tid = threadIdx.x;
  // global index of data element in data block
  int idx = blockIdx.x * blockDim.x * 8 + tid;

  // convert global data pointer to the local pointer of this block
  int *idata = g_idata + blockIdx.x * blockDim.x * 8;

  // unrolling 8
  if (idx + 7 * blockDim.x < n)
  {
    int a1 = g_idata[idx];
    int a2 = g_idata[idx + blockDim.x];
    int a3 = g_idata[idx + 2 * blockDim.x];
    int a4 = g_idata[idx + 3 * blockDim.x];
    int b1 = g_idata[idx + 4 * blockDim.x];
    int b2 = g_idata[idx + 5 * blockDim.x];
    int b3 = g_idata[idx + 6 * blockDim.x];
    int b4 = g_idata[idx + 7 * blockDim.x];
    g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
  }
  __syncthreads();

  // in-place reduction and complete unroll
  if (iBlockSize >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
  __syncthreads();
  if (iBlockSize >= 512 && tid < 256)  idata[tid] += idata[tid + 256];
  __syncthreads();
  if (iBlockSize >= 256 && tid < 128)  idata[tid] += idata[tid + 128];
  __syncthreads();
  if (iBlockSize >= 128 && tid < 64)   idata[tid] += idata[tid + 64];
  __syncthreads();

  // unrolling warp
  if (tid < 32)
  {
    volatile int *vsmem = idata;
    vsmem[tid] += vsmem[tid + 32];
    vsmem[tid] += vsmem[tid + 16];
    vsmem[tid] += vsmem[tid + 8];
    vsmem[tid] += vsmem[tid + 4];
    vsmem[tid] += vsmem[tid + 2];
    vsmem[tid] += vsmem[tid + 1];
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int recursiveReduce(int *data, int const size)
{
  if (size == 1)
    return data[0];
  int const stride = size / 2;

  for (int i = 0; i < stride; i++)
  {
    data[i] += data[i + stride];
  }

  return recursiveReduce(data, stride);
}

int main(int argc, char **argv)
{
  int size = 1 << 24;
  printf("With array size %d ", size);

  int blocksize = 512;
  if (argc > 1)
  {
    blocksize = atoi(argv[1]);
  }

  dim3 block(blocksize, 1);
  dim3 grid((size + block.x - 1) / block.x, 1);
  printf("grid %d block %d\n", grid.x, block.x);

  // allocate host memory
  size_t bytes = size * sizeof(int);
  int *h_idata = (int *)malloc(bytes);
  int *h_odata = (int *)malloc(grid.x * sizeof(int));
  int *tmp = (int *)malloc(bytes);

  // initialize the array
  for (int i = 0; i < size; i++)
  {
    // mask off high 2 bytes to force max number to 255
    h_idata[i] = (int)(rand() & 0xFF);
  }
  memcpy(tmp, h_idata, bytes);

  clock_t iStart, iElaps;
  int gpu_sum = 0;

  // allocate device memory
  int *d_idata = NULL;
  int *d_odata = NULL;
  hipMalloc((void **)&d_idata, bytes);
  hipMalloc((void **)&d_odata, grid.x * sizeof(int));

  // cpu reduction
  iStart = clock();
  int cpu_sum = recursiveReduce(tmp, size);
  iElaps = (long int)(clock() - iStart);
  printf("cpu reduce elapsed %ld ms cpu_sum: %d\n", iElaps, cpu_sum);

  // kernel 1: reduceNeighbored
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu Neighbored elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x, block.x);

  // kernel 2: reduceNeighboredLess
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceNeighboredLess<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu NeighboredL elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x, block.x);

  // kernel 3: reduceInterLeaved
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x; i++)
    gpu_sum += h_odata[i];
  printf("gpu Interleaved elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x, block.x);

  // kernel 4: reduceUnrolling2
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceUnrolling2<<<grid.x / 2, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x / 2 * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x / 2; i++)
    gpu_sum += h_odata[i];
  printf("gpu unroll2 elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x / 2, block.x);

  // kernel 5: reduceUnrolling4
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceUnrolling4<<<grid.x / 4, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x / 4; i++)
    gpu_sum += h_odata[i];
  printf("gpu unroll4 elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x / 4, block.x);

  // kernel 5: reducecompleteUnrolling4
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceCompleteUnrollWarp8<<<grid.x / 8, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x / 8; i++)
    gpu_sum += h_odata[i];
  printf("gpu complete unroll elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x / 8, block.x);

  // kernel 6: reducecompleteUnrolling4
  hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  iStart = clock();
  reduceCompleteUnrollTemplate<256><<<grid.x / 8, block>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  iElaps = (long int)(clock() - iStart);
  hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost);
  gpu_sum = 0;
  for (int i = 0; i < grid.x / 8; i++)
    gpu_sum += h_odata[i];
  printf("gpu template complete unroll elapsed %ld ms gpu_sum: %d <<<grid %d block %d>>>\n",
         iElaps, gpu_sum, grid.x / 8, block.x);

  free(h_idata);
  free(h_odata);
  hipFree(d_idata);
  hipFree(d_odata);
  hipDeviceReset();

  bool bResult = (gpu_sum == cpu_sum);
  if (!bResult)
    printf("Test failed!\n");
  else
  {
    printf("cpu_sum = gpu_sum\n");
  }
  return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#define CUDA_CHECK(call) { 
    hipError_t err = call;
    if (err != hipSuccess) { 
        printf("Error: %s:%d, ", __FILE__, __LINE__); 
        printf("code: %d, reason: %s\n", err, hipGetErrorString(err)); 
        exit(1); 
    } 
}

__global__ void print_hello()
{
    printf("Hello Cu\n");
}

int main()
{
    print_hello<<<1, 10>>>();
    CUDA_CHECK(hipDeviceSynchronize());
    // hipDeviceReset();
    return 0;
}




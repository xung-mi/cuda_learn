#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int *d_ptr;
    hipError_t err = hipMalloc((void**)&d_ptr, -1); // Cấp phát kích thước âm -> lỗi

    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    return 0;
}


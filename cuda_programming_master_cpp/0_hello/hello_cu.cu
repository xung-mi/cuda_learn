#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void print_hello(){
    printf("Hello Cu\n");
}

int main(){
    print_hello<<<1,10>>>();
    hipDeviceSynchronize();
    // cudaDeviceReset();
    return 0;
}

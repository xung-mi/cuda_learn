
#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void print_threadIds(){
       printf("threadIdx.x: %d, threadIdx.y : %d, threadIdx.z : %d\n", 
              threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_blockIds(){
       printf("blockIdx.x: %d, blockIdx.y : %d, blockIdx.z : %d\n", 
              blockIdx.x, blockIdx.y, blockIdx.z);
}

__global__ void print_blockDim(){
       printf("blockDim.x: %d, blockDim.y : %d, blockDim.z : %d\n", 
              blockDim.x, blockDim.y, blockDim.z);
}

__global__ void print_gridDim(){
       printf("gridDim.x: %d, gridDim.y : %d, gridDim.z : %d\n", 
              gridDim.x, gridDim.y, gridDim.z);
}

int main() {
    int nx, ny;
    nx = 8;
    ny = 8;

    dim3 block(4,4);
    dim3 grid(nx/block.x, ny/block.y);

    print_gridDim<<<grid, block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}

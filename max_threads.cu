#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Lấy thông tin GPU device 0

    printf("Device name: %s\n", prop.name);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Max blocks per dimension: (%d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Max threads per block dimension: (%d, %d, %d)\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);

    return 0;
}

